#include "hip/hip_runtime.h"
#pragma once
#ifndef _DEBUG
#define _DEBUG
#endif

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "foundation\PxVec2.h"
#include "foundation\PxVec3.h"
#include "foundation\PxVec4.h"
#include ""
//
using namespace physx;

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
//TODO: Take in bitmap and check validity of each particle
//TODO: Pass in the GL Buffer for colour and Emiiter position and modify colour based on distance
extern "C"__global__ void UpdateParticlesKernel(PxVec3* destPositions, PxVec4* srcPositions)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(srcPositions[i].magnitude() > 25.0f)
	{
		srcPositions[i].x = 0;		
		srcPositions[i].y = 0;	
		srcPositions[i].z = 0;		
	}
	
	destPositions[i].x = srcPositions[i].x;		
	destPositions[i].y = srcPositions[i].y;	
	destPositions[i].z = srcPositions[i].z;	
}

// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void updateBillboardVB(
	PxVec3* destPositions,
	PxVec4* srcPositions, 
	PxU32* validParticleBitmap,
	PxU32 validParticleRange,
	PxU32 maxParticles)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*sizeof(PxVec3);

				// copy position
				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				++destIndex;
			}
		}
	}
}

// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void UpdateColourSystem(
	PxVec3* destPositions,
	PxVec4* srcPositions, 
	PxU32* validParticleBitmap,
	float* srcLifetimes,
	float initialLifetime,
	PxVec4* colours,
	PxVec4 startColour,
	PxVec4 endColour,
	PxVec4 fadeRate,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*sizeof(PxVec3);

				const PxU32 lifeOffset = destIndex * sizeof(float);

				// copy position
				PxVec3* p = ptrOffset(destPositions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				float* l = ptrOffset(srcLifetimes, lifeOffset);
				float percent = (*l / initialLifetime);

				const PxU32 colourOffset = destIndex * sizeof(PxVec4);

				PxVec4* c = ptrOffset(colours, colourOffset);
				
				c->x = (startColour.x - endColour.x) * percent;
				c->y = (startColour.y - endColour.y) * percent;
				c->z = (startColour.z - endColour.z) * percent;
				c->w = (startColour.w - endColour.w) * percent;

				++destIndex;
			}
		}
	}
}