#include "hip/hip_runtime.h"
#pragma once
#ifndef _DEBUG
#define _DEBUG
#endif

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "foundation\PxVec2.h"
#include "foundation\PxVec3.h"
#include "foundation\PxVec4.h"
#include ""
//
using namespace physx;

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
//TODO: Take in bitmap and check validity of each particle
//TODO: Pass in the GL Buffer for colour and Emiiter position and modify colour based on distance
extern "C"__global__ void UpdateParticlesKernel(PxVec3* destPositions, PxVec4* srcPositions)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(srcPositions[i].magnitude() > 25.0f)
	{
		srcPositions[i].x = 0;		
		srcPositions[i].y = 0;	
		srcPositions[i].z = 0;		
	}
	
	destPositions[i].x = srcPositions[i].x;		
	destPositions[i].y = srcPositions[i].y;	
	destPositions[i].z = srcPositions[i].z;	
}

// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void updateBillboardVB(
	PxVec3* destPositions,
	const PxVec4* srcPositions, 
	float* lifetimes,
	const PxU32* validParticleBitmap,
	PxU32 validParticleRange)
{
	for (int i = threadIdx.x; i < validParticleRange; i+=blockDim.x)
	{
		destPositions[i].x = srcPositions[i].x;
		destPositions[i].y = srcPositions[i].y;
		destPositions[i].z = srcPositions[i].z;
	}

	for (int i = threadIdx.x; i < 100000; i+=blockDim.x)
	{
		if(lifetimes[i] <= 0)
		{
			lifetimes[i] = 0.0f;
			destPositions[i].x = 0.0f;
			destPositions[i].y = 0.0f;
			destPositions[i].z = 0.0f;
		}
	}
	//if (!threadIdx.x)
	//	gOffset = 0;

	//__syncthreads();

	//if (validParticleRange)
	//{
	//	for (PxU32 w=threadIdx.x; w <= (validParticleRange) >> 5; w+=blockDim.x)
	//	{
	//		const PxU32 srcBaseIndex = w << 5;

	//		// reserve space in the output vertex buffer based on
	//		// population count of validity bitmap (avoids excess atomic ops)
	//		PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

	//		for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
	//		{
	//			PxU32 index = srcBaseIndex | __ffs(b)-1;

	//			const PxU32 offset = destIndex*12;

	//			// copy position
	//			PxVec3* p = ptrOffset(destPositions, offset);
	//			p->x = srcPositions[index].x;
	//			p->y = srcPositions[index].y;
	//			p->z = srcPositions[index].z;

	//			/*if(srcPositions[index].x >= 12)
	//				srcLifetimes[index] = 0.0f;*/

	//			++destIndex;
	//		}
	//	}
	//}
}

