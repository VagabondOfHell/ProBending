#include "hip/hip_runtime.h"
#pragma once
#ifndef _DEBUG
#define _DEBUG
#endif

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "foundation\PxVec2.h"
#include "foundation\PxVec3.h"
#include "foundation\PxVec4.h"
#include ""
//
using namespace physx;

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
//TODO: Take in bitmap and check validity of each particle
//TODO: Pass in the GL Buffer for colour and Emiiter position and modify colour based on distance
extern "C"__global__ void UpdateParticlesKernel(PxVec3* destPositions, PxVec4* srcPositions)
{
	int i =  blockIdx.x * blockDim.x + threadIdx.x;
	
	if(srcPositions[i].magnitude() > 25.0f)
	{
		srcPositions[i].x = 0;		
		srcPositions[i].y = 0;	
		srcPositions[i].z = 0;		
	}
	
	destPositions[i].x = srcPositions[i].x;		
	destPositions[i].y = srcPositions[i].y;	
	destPositions[i].z = srcPositions[i].z;	
}

