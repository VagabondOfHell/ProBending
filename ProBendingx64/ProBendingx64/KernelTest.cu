#include "hip/hip_runtime.h"
#pragma once
#ifndef _DEBUG
#define _DEBUG
#endif

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "AffectorParameters.h"

#include ""

using namespace physx;

template <typename T>
__device__ T* ptrOffset(T* p, PxU32 byteOffset)
{
	return (T*)((unsigned char*)(p) + byteOffset);
}

#if __CUDA_ARCH__ < 200
__device__ PxU32 gOffset;
#else
__device__ __shared__ PxU32 gOffset;
#endif

// copies positions and alpha to the destination vertex buffer based on 
// validity bitmap and particle life times
extern "C" __global__ void UpdateParticleSystem(
	GPUResourcePointers graphicsResources,
	PxVec4* srcPositions, 
	PxU32* validParticleBitmap,
	float* srcLifetimes,
	float initialLifetime,
	GPUParamsCollection* affectorParams,
	PxU32 validParticleRange)
{
	if (!threadIdx.x)
		gOffset = 0;

	__syncthreads();

	if (validParticleRange)
	{
		for (PxU32 w=threadIdx.x; w <= (validParticleRange - 1) >> 5; w+=blockDim.x)
		{
			const PxU32 srcBaseIndex = w << 5;

			// reserve space in the output vertex buffer based on
			// population count of validity bitmap (avoids excess atomic ops)
			PxU32 destIndex = atomicAdd(&gOffset, __popc(validParticleBitmap[w]));

			for (PxU32 b=validParticleBitmap[w]; b; b &= b-1) 
			{
				PxU32 index = srcBaseIndex | __ffs(b)-1;

				const PxU32 offset = destIndex*sizeof(PxVec4);

				// copy position, save the w
				PxVec4* p = ptrOffset(graphicsResources.positions, offset);
				p->x = srcPositions[index].x;
				p->y = srcPositions[index].y;
				p->z = srcPositions[index].z;

				//If there are any affectors to apply
				if(affectorParams)
				{
					const PxU32 lifeOffset = index * sizeof(float);
					float* l = ptrOffset(srcLifetimes, lifeOffset);
					float percent = (*l / initialLifetime);

					if(affectorParams->colourFadeParams)
					{
						//Need to use colours
						PxVec4* c = ptrOffset(graphicsResources.primaryColour, offset);

						c->x = affectorParams->colourFadeParams->endColour.x - (affectorParams->colourFadeParams->colourDifference.x * percent);
						c->y = affectorParams->colourFadeParams->endColour.y - (affectorParams->colourFadeParams->colourDifference.y * percent);
						c->z = affectorParams->colourFadeParams->endColour.z - (affectorParams->colourFadeParams->colourDifference.z * percent);
						c->w = affectorParams->colourFadeParams->endColour.w - (affectorParams->colourFadeParams->colourDifference.w * percent);
					}

					if(affectorParams->scaleParameters)
					{
						if(affectorParams->scaleParameters->enlarge)
							p->w = affectorParams->scaleParameters->maxScale - (affectorParams->scaleParameters->scaleDiff * percent);
						else
							p->w = affectorParams->scaleParameters->minScale + (affectorParams->scaleParameters->scaleDiff * percent);
					}
				}
				
				++destIndex;
			}
		}
	}
}